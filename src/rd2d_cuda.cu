 /**************************************************************************
 *   This file is part of LaFluxxy                                        *
 *   https://github.com/imc-codeteam/lafluxxy                             *
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   LaFluxxy is free software: you can redistribute it and/or modify     *
 *   it under the terms of the GNU General Public License as published    *
 *   by the Free Software Foundation, either version 3 of the License,    *
 *   or (at your option) any later version.                               *
 *                                                                        *
 *   LaFluxxy is distributed in the hope that it will be useful,          *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

// add auxiliary cuda functions
#include "check_cuda.h"
#include "cuda_events.h"

// include kernels
#include "kernels/laplacians.h"
#include "kernels/reaction_gray_scott.h"
#include "kernels/update.h"

// other includes
#include "rd2d_cuda.h"

RD2D_CUDA::RD2D_CUDA() {

}

/**
 * @brief      Run time-integration on GPU
 */
void RD2D_CUDA::update_step() {
    std::cout << "Starting time-integration" << std::endl;

    std::cout << "Setting grids and blocks...           ";
    dim3 gridx(this->my / this->pencils, 1, 1);
    dim3 blockx(this->mx, this->pencils, 1);
    dim3 gridy(this->mx / this->pencils, 1, 1);
    dim3 blocky(this->pencils, this->my, 1);
    unsigned int block = this->mx;
    unsigned int grid = (this->ncells + this->mx - 1) / this->mx;

    unsigned shared_mem_size = 0;
    if(this->zeroflux) {
        shared_mem_size = this->pencils * this->mx * sizeof(float);
    } else {
        shared_mem_size = this->pencils * (this->mx + 2) * sizeof(float);
    }
    std::cout << donestring << std::endl << std::endl;

    // keep track of time
    hipEvent_t startEvent, stopEvent;
    hipEvent_t startEventKernel, stopEventKernel;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    checkCuda( hipEventCreate(&startEventKernel) );
    checkCuda( hipEventCreate(&stopEventKernel) );

    // keep track of time
    float milliseconds = 0;
    float laplacian_x_times = 0;
    float laplacian_y_times = 0;
    float reaction_times = 0;
    float update_times = 0;
    float laplacian_summation_times = 0;

    start_event(&startEvent);

    for(unsigned int i=0; i<this->tsteps; i++) {

        // calculate laplacian for A
        if(this->zeroflux) {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_zeroflux<<<gridx,blockx,shared_mem_size>>>(d_a, d_dx2);
            laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_zeroflux<<<gridy,blocky,shared_mem_size>>>(d_a, d_dy2);
            laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);
        } else {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_pbc<<<gridx,blockx,shared_mem_size>>>(d_a, d_dx2);
            laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_pbc<<<gridy,blocky,shared_mem_size>>>(d_a, d_dy2);
            laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        }

        // sum all three derivative components
        start_event(&startEventKernel);
        construct_laplacian_a<<<grid,block>>>(d_da, d_dx2, d_dy2);
        laplacian_summation_times += stop_event(&startEventKernel, &stopEventKernel);;

        // calculate laplacian for B
        if(this->zeroflux) {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_zeroflux<<<gridx,blockx,shared_mem_size>>>(d_b, d_dx2);
            laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_zeroflux<<<gridy,blocky,shared_mem_size>>>(d_b, d_dy2);
            laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        } else {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_pbc<<<gridx,blockx,shared_mem_size>>>(d_b, d_dx2);
            laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_pbc<<<gridy,blocky,shared_mem_size>>>(d_b, d_dy2);
            laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        }

        // sum all three derivative components
        start_event(&startEventKernel);
        construct_laplacian_b<<<grid,block>>>(d_db, d_dx2, d_dy2);
        laplacian_summation_times += stop_event(&startEventKernel, &stopEventKernel);;

        // calculate reaction
        start_event(&startEventKernel);
        reaction_gray_scott<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
        reaction_times += stop_event(&startEventKernel, &stopEventKernel);;

        // update
        start_event(&startEventKernel);
        update<<<grid,block>>>(d_a, d_b, d_da, d_db, d_ra, d_rb);
        update_times += stop_event(&startEventKernel, &stopEventKernel);;;
    }

    // stop timer
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&milliseconds, startEvent, stopEvent) );

    // copy results back
    int bytes = this->ncells * sizeof(float);
    checkCuda( hipMemcpy(this->a, this->d_a, bytes, hipMemcpyDeviceToHost) );
    checkCuda( hipMemcpy(this->b, this->d_b, bytes, hipMemcpyDeviceToHost) );

    float other_times = milliseconds - laplacian_x_times - laplacian_y_times -
                        reaction_times - update_times - laplacian_summation_times;

    printf("------------------------------------------\n");
    printf("  Integration time: %12.6f ms\n", milliseconds);
    printf("  Laplacian x:      %12.6f ms\n", laplacian_x_times);
    printf("  Laplacian y:      %12.6f ms\n", laplacian_y_times);
    printf("  Laplacian sum:    %12.6f ms\n", laplacian_summation_times);
    printf("  Reaction:         %12.6f ms\n", reaction_times);
    printf("  Time step:        %12.6f ms\n", update_times);
    printf("  Other:            %12.6f ms\n\n", other_times);
    printf("------------------------------------------\n");
    printf("\n");

    // clean up
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );

    std::cout << std::endl;
}

/**
 * @brief      Initialize all variables
 */
void RD2D_CUDA::initialize_variables(const std::vector<float>& _a, const std::vector<float>& _b) {
    std::cout << "Loading device variables." << std::endl;

    // build initial concentrations
    std::cout << "Constructing initial concentrations...";
    this->a = new float[this->ncells];
    this->b = new float[this->ncells];
    memcpy(this->a, &_a[0], sizeof(float) * _a.size());
    memcpy(this->b, &_b[0], sizeof(float) * _b.size());
    std::cout << donestring << std::endl;

    // allocate size on device
    std::cout << "Allocating variables on GPU device... ";
    int bytes = this->ncells * sizeof(float);
    checkCuda( hipMalloc((void**)&this->d_a, bytes) );
    checkCuda( hipMalloc((void**)&this->d_b, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dx2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dy2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_ra, bytes) );
    checkCuda( hipMalloc((void**)&this->d_rb, bytes) );
    checkCuda( hipMalloc((void**)&this->d_da, bytes) );
    checkCuda( hipMalloc((void**)&this->d_db, bytes) );
    std::cout << donestring << std::endl;

    // copy data to device
    std::cout << "Copying data to GPU device...         ";
    checkCuda( hipMemcpy(this->d_a, this->a, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(this->d_b, this->b, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemset(this->d_dx2, 0, bytes) );
    checkCuda( hipMemset(this->d_dy2, 0, bytes) );
    checkCuda( hipMemset(this->d_ra, 0, bytes) );
    checkCuda( hipMemset(this->d_rb, 0, bytes) );
    checkCuda( hipMemset(this->d_da, 0, bytes) );
    checkCuda( hipMemset(this->d_db, 0, bytes) );
    std::cout << donestring << std::endl;

    // set constants
    std::cout << "Setting constant variables on GPU...  ";
    float _diffcon_a = this->Da / (this->dx * this->dx);
    float _diffcon_b = this->Db / (this->dx * this->dx);
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_diffcon_a), &_diffcon_a, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_diffcon_b), &_diffcon_b, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &this->dt, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_mx), &this->mx, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_my), &this->my, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_pencils), &this->pencils, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_ncells), &this->ncells, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_f), &this->f, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_k), &this->k, sizeof(float)) );
    std::cout << donestring << std::endl;

    std::cout << "All ready for time-integration." << std::endl << std::endl;
}

/**
 * @brief      Clean-up all variables
 */
void RD2D_CUDA::cleanup_variables() {
    std::cout << "Cleaning Integration variables...     ";
    checkCuda( hipFree(this->d_a) );
    checkCuda( hipFree(this->d_b) );
    checkCuda( hipFree(this->d_ra) );
    checkCuda( hipFree(this->d_rb) );
    checkCuda( hipFree(this->d_da) );
    checkCuda( hipFree(this->d_db) );
    checkCuda( hipFree(this->d_dx2) );
    checkCuda( hipFree(this->d_dy2) );

    delete [] this->a;
    delete [] this->b;

    std::cout << donestring << std::endl;
    std::cout << std::endl;
}
