 /**************************************************************************
 *   This file is part of LaFluxxy                                        *
 *   https://github.com/imc-codeteam/lafluxxy                             *
 *                                                                        *
 *   Author: Ivo Filot <i.a.w.filot@tue.nl>                               *
 *                                                                        *
 *   LaFluxxy is free software: you can redistribute it and/or modify     *
 *   it under the terms of the GNU General Public License as published    *
 *   by the Free Software Foundation, either version 3 of the License,    *
 *   or (at your option) any later version.                               *
 *                                                                        *
 *   LaFluxxy is distributed in the hope that it will be useful,          *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty          *
 *   of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.              *
 *   See the GNU General Public License for more details.                 *
 *                                                                        *
 *   You should have received a copy of the GNU General Public License    *
 *   along with this program.  If not, see http://www.gnu.org/licenses/.  *
 *                                                                        *
 **************************************************************************/

#include <array>

// add auxiliary cuda functions
#include "check_cuda.h"
#include "cuda_events.h"

// include kernels
#include "kernels/laplacians.h"
#include "kernels/reaction_kinetics.h"
#include "kernels/update.h"

// other includes
#include "rd2d_cuda.h"

RD2D_CUDA::RD2D_CUDA() {

}

/**
 * @brief      Run time-integration on GPU
 */
void RD2D_CUDA::update_step() {
    // std::cout << "Starting time-integration" << std::endl;

    // std::cout << "Setting grids and blocks...           ";
    dim3 gridx(this->my / this->pencils, 1, 1);
    dim3 blockx(this->mx, this->pencils, 1);
    dim3 gridy(this->mx / this->pencils, 1, 1);
    dim3 blocky(this->pencils, this->my, 1);
    unsigned int block = this->mx;
    unsigned int grid = (this->ncells + this->mx - 1) / this->mx;

    unsigned shared_mem_size = 0;
    if(this->zeroflux) {
        shared_mem_size = this->pencils * this->mx * sizeof(float);
    } else {
        shared_mem_size = this->pencils * (this->mx + 2) * sizeof(float);
    }
    // std::cout << donestring << std::endl << std::endl;

    // keep track of time
    hipEvent_t startEvent, stopEvent;
    hipEvent_t startEventKernel, stopEventKernel;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    checkCuda( hipEventCreate(&startEventKernel) );
    checkCuda( hipEventCreate(&stopEventKernel) );

    // keep track of time
    this->milliseconds = 0;
    this->laplacian_x_times = 0;
    this->laplacian_y_times = 0;
    this->reaction_times = 0;
    this->update_times = 0;
    this->laplacian_summation_times = 0;

    start_event(&startEvent);

    for(unsigned int i=0; i<this->tsteps; i++) {

        // calculate laplacian for A
        if(this->zeroflux) {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_zeroflux<<<gridx,blockx,shared_mem_size>>>(d_a, d_dx2);
            this->laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_zeroflux<<<gridy,blocky,shared_mem_size>>>(d_a, d_dy2);
            this->laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);
        } else if(this->has_mask) {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_masked<<<gridx,blockx,shared_mem_size>>>(d_a, d_mask, d_dx2);
            this->laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_masked<<<gridy,blocky,shared_mem_size>>>(d_a, d_mask, d_dy2);
            this->laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        } else {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_pbc<<<gridx,blockx,shared_mem_size>>>(d_a, d_dx2);
            this->laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_pbc<<<gridy,blocky,shared_mem_size>>>(d_a, d_dy2);
            this->laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        }

        // sum all three derivative components
        start_event(&startEventKernel);
        construct_laplacian_a<<<grid,block>>>(d_da, d_dx2, d_dy2);
        this->laplacian_summation_times += stop_event(&startEventKernel, &stopEventKernel);;

        // calculate laplacian for B
        if(this->zeroflux) {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_zeroflux<<<gridx,blockx,shared_mem_size>>>(d_b, d_dx2);
            this->laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_zeroflux<<<gridy,blocky,shared_mem_size>>>(d_b, d_dy2);
            this->laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        } else if(this->has_mask) {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_masked<<<gridx,blockx,shared_mem_size>>>(d_b, d_mask, d_dx2);
            this->laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_masked<<<gridy,blocky,shared_mem_size>>>(d_b, d_mask, d_dy2);
            this->laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        } else {
            // x2 derivative
            start_event(&startEventKernel);
            derivative_x2_pbc<<<gridx,blockx,shared_mem_size>>>(d_b, d_dx2);
            this->laplacian_x_times += stop_event(&startEventKernel, &stopEventKernel);;

            // y2 derivative
            start_event(&startEventKernel);
            derivative_y2_pbc<<<gridy,blocky,shared_mem_size>>>(d_b, d_dy2);
            this->laplacian_y_times += stop_event(&startEventKernel, &stopEventKernel);;
        }

        // sum all derivative components
        start_event(&startEventKernel);
        construct_laplacian_b<<<grid,block>>>(d_db, d_dx2, d_dy2);
        this->laplacian_summation_times += stop_event(&startEventKernel, &stopEventKernel);;

        // calculate reaction
        start_event(&startEventKernel);

        // execute reaction kinetics step
        switch(this->reacttype) {
            case KINETICS::GRAY_SCOTT:
                reaction_gray_scott<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
            break;
            case KINETICS::BRUSSELATOR:
                reaction_brusselator<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
            break;
            case KINETICS::BARKLEY:
                reaction_barkley<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
            break;
            case KINETICS::LOTKA_VOLTERRA:
                reaction_lotka_volterra<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
            break;
            case KINETICS::FITZHUGH_NAGUMO:
                reaction_fitzhugh_nagumo<<<grid,block>>>(d_a, d_b, d_ra, d_rb);
            break;
            default:
                throw std::runtime_error("Invalid reaction type");
        }

        this->reaction_times += stop_event(&startEventKernel, &stopEventKernel);;

        // update
        start_event(&startEventKernel);
        update<<<grid,block>>>(d_a, d_b, d_da, d_db, d_ra, d_rb);
        this->update_times += stop_event(&startEventKernel, &stopEventKernel);;;
    }

    // stop timer
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&this->milliseconds, startEvent, stopEvent) );

    // copy results back
    int bytes = this->ncells * sizeof(float);
    checkCuda( hipMemcpy(this->a, this->d_a, bytes, hipMemcpyDeviceToHost) );
    checkCuda( hipMemcpy(this->b, this->d_b, bytes, hipMemcpyDeviceToHost) );

    this->other_times = this->milliseconds - this->laplacian_x_times - this->laplacian_y_times -
                        this->reaction_times - this->update_times - this->laplacian_summation_times;

    // printf("------------------------------------------\n");
    // printf("  Integration time: %12.6f ms\n", this->milliseconds);
    // printf("  Laplacian x:      %12.6f ms\n", this->laplacian_x_times);
    // printf("  Laplacian y:      %12.6f ms\n", this->laplacian_y_times);
    // printf("  Laplacian sum:    %12.6f ms\n", this->laplacian_summation_times);
    // printf("  Reaction:         %12.6f ms\n", this->reaction_times);
    // printf("  Time step:        %12.6f ms\n", this->update_times);
    // printf("  Other:            %12.6f ms\n\n", this->other_times);
    // printf("------------------------------------------\n");
    // printf("\n");

    // clean up
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );
}

/**
 * @brief      Initialize all variables
 */
void RD2D_CUDA::initialize_variables(const std::vector<float>& _a,
                                     const std::vector<float>& _b,
                                     const std::vector<float>& _mask) {
    // std::cout << "Loading device variables." << std::endl;

    // build initial concentrations
    // std::cout << "Constructing initial concentrations...";
    this->a = new float[this->ncells];
    this->b = new float[this->ncells];
    this->mask = new float[this->ncells];
    memcpy(this->a, &_a[0], sizeof(float) * _a.size());
    memcpy(this->b, &_b[0], sizeof(float) * _b.size());
    memcpy(this->mask, &_mask[0], sizeof(float) * _mask.size());
    // std::cout << donestring << std::endl;

    // allocate size on device
    // std::cout << "Allocating variables on GPU device... ";
    int bytes = this->ncells * sizeof(float);
    checkCuda( hipMalloc((void**)&this->d_a, bytes) );
    checkCuda( hipMalloc((void**)&this->d_b, bytes) );
    checkCuda( hipMalloc((void**)&this->d_mask, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dx2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_dy2, bytes) );
    checkCuda( hipMalloc((void**)&this->d_ra, bytes) );
    checkCuda( hipMalloc((void**)&this->d_rb, bytes) );
    checkCuda( hipMalloc((void**)&this->d_da, bytes) );
    checkCuda( hipMalloc((void**)&this->d_db, bytes) );
    // std::cout << donestring << std::endl;

    // copy data to device
    // std::cout << "Copying data to GPU device...         ";
    checkCuda( hipMemcpy(this->d_a, this->a, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(this->d_b, this->b, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(this->d_mask, this->mask, bytes, hipMemcpyHostToDevice) );
    checkCuda( hipMemset(this->d_dx2, 0, bytes) );
    checkCuda( hipMemset(this->d_dy2, 0, bytes) );
    checkCuda( hipMemset(this->d_ra, 0, bytes) );
    checkCuda( hipMemset(this->d_rb, 0, bytes) );
    checkCuda( hipMemset(this->d_da, 0, bytes) );
    checkCuda( hipMemset(this->d_db, 0, bytes) );
    // std::cout << donestring << std::endl;

    // set constants
    // std::cout << "Setting constant variables on GPU...  ";
    float _diffcon_a = this->Da / (this->dx * this->dx);
    float _diffcon_b = this->Db / (this->dx * this->dx);
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_diffcon_a), &_diffcon_a, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_diffcon_b), &_diffcon_b, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &this->dt, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_mx), &this->mx, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_my), &this->my, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_pencils), &this->pencils, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_ncells), &this->ncells, sizeof(unsigned int)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_c1), &this->c1, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_c2), &this->c2, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_c3), &this->c3, sizeof(float)) );
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_c4), &this->c4, sizeof(float)) );
    // std::cout << donestring << std::endl;

    // std::cout << "All ready for time-integration." << std::endl << std::endl;
}

/**
 * @brief      Clean-up all variables
 */
void RD2D_CUDA::cleanup_variables() {
    // std::cout << "Cleaning Integration variables...     ";
    checkCuda( hipFree(this->d_a) );
    checkCuda( hipFree(this->d_b) );
    checkCuda( hipFree(this->d_mask) );
    checkCuda( hipFree(this->d_ra) );
    checkCuda( hipFree(this->d_rb) );
    checkCuda( hipFree(this->d_da) );
    checkCuda( hipFree(this->d_db) );
    checkCuda( hipFree(this->d_dx2) );
    checkCuda( hipFree(this->d_dy2) );

    delete [] this->a;
    delete [] this->b;
    delete [] this->mask;

    // std::cout << donestring << std::endl;
    // std::cout << std::endl;
}
